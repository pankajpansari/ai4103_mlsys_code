#include "hip/hip_runtime.h"
#include "blur.cuh"
#include "ErrorCheck.cuh"
#include <cstdio>
#include <hip/hip_runtime.h>

__device__
void cuda_blur_kernel_convolution(uint raw_data_index, const float* gpu_raw_data,
                                  const float* gpu_blur_v, float* gpu_out_data,
                                  const unsigned int n_frames,
                                  const unsigned int blur_v_size) {
  if (raw_data_index < blur_v_size) {
    for (int j = 0; j <= raw_data_index; j++)
      gpu_out_data[raw_data_index] += gpu_raw_data[raw_data_index - j] * gpu_blur_v[j]; 
  } else if (raw_data_index >= blur_v_size && raw_data_index < n_frames) {
    for (int j = 0; j < blur_v_size; j++)
      gpu_out_data[raw_data_index] += gpu_raw_data[raw_data_index - j] * gpu_blur_v[j]; 
    }

}

__global__
void cuda_blur_kernel(const float *gpu_raw_data, const float *gpu_blur_v,
                      float *gpu_out_data, int n_frames, int blur_v_size) {
    uint raw_data_index = blockIdx.x * blockDim.x + threadIdx.x;

    while (raw_data_index < n_frames) {
        cuda_blur_kernel_convolution(raw_data_index, gpu_raw_data,
                                     gpu_blur_v, gpu_out_data,
                                     n_frames, blur_v_size);
        raw_data_index += gridDim.x * blockDim.x;
    }
}


void cuda_call_blur_kernel(const unsigned int blocks,
                            const unsigned int threads_per_block,
                            const float *raw_data,
                            const float *blur_v,
                            float *out_data,
                            const unsigned int n_frames,
                            const unsigned int blur_v_size) {
    float* gpu_raw_data;
    hipMalloc(&gpu_raw_data, n_frames * sizeof(float));

    hipMemcpy(gpu_raw_data, raw_data, n_frames * sizeof(float), hipMemcpyHostToDevice);
    float* gpu_blur_v;
    hipMalloc(&gpu_blur_v, blur_v_size * sizeof(float));

    hipMemcpy(gpu_blur_v, blur_v, blur_v_size * sizeof(float), hipMemcpyHostToDevice);

    float* gpu_out_data;
    hipMalloc(&gpu_out_data, n_frames * sizeof(float));
    hipMemset(gpu_out_data, 0, n_frames * sizeof(float));
    
    cuda_blur_kernel<<<(int) blocks, (int) threads_per_block>>>(gpu_raw_data, gpu_blur_v, gpu_out_data, n_frames, blur_v_size);


    hipMemcpy(out_data, gpu_out_data, n_frames * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(gpu_raw_data);
    hipFree(gpu_blur_v);
    hipFree(gpu_out_data); 

}
